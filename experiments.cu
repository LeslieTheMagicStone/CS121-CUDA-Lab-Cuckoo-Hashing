#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <cmath>
#include <fstream>
#include "parallel.cuh"
#include "vanilla.hpp"

using namespace std;
using namespace chrono;

void generateRandomKeys(vector<uint32_t> &keys, uint32_t n)
{
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<uint32_t> dis(1, UINT32_MAX);

    for (uint32_t i = 0; i < n; ++i)
    {
        keys.push_back(dis(gen));
    }
}

void readKeysFromFile(const string &filename, vector<uint32_t> &keys)
{
    ifstream file(filename);
    uint32_t key;
    while (file >> key)
    {
        keys.push_back(key);
    }
}

void simpleDemo()
{
    uint32_t n = 8;
    uint32_t t = 2;
    uint32_t size = 10;
    uint32_t maxIter = 4 * log2(n);
    vector<uint32_t> keys = {1, 2, 3, 4, 5, 6, 7, 8};

    std::cout << "Running simple demo with n=8, t=2, size=10" << std::endl;

    uint32_t rehashesSeq = 0;
    SequentialHash sh(size, t, maxIter);
    sh.insertKeys(keys.data(), n, rehashesSeq);

    std::cout << "Final table:" << std::endl;
    sh.printTables();

    uint32_t rehashesPar = 0;
    ParallelHash ph(size, t, maxIter);
    ph.insertKeys(keys.data(), n, rehashesPar);

    std::cout << "Final table:" << std::endl;
    ph.printTables();
}

void experiment1(uint32_t t)
{
    for (uint32_t exp = 10; exp <= 24; ++exp)
    {
        for (int iter = 0; iter < 5; iter++)
        {
            uint32_t n = pow(2, exp);
            uint32_t size = pow(2, 25);
            uint32_t maxIter = 4 * log2(n);
            vector<uint32_t> keys;
            generateRandomKeys(keys, n);

            uint32_t rehashesSeq = 0;
            SequentialHash sh(size, t, maxIter);
            auto start = high_resolution_clock::now();
            sh.insertKeys(keys.data(), n, rehashesSeq);
            auto end = high_resolution_clock::now();
            auto durationSeq = duration_cast<microseconds>(end - start).count();

            uint32_t rehashesPar = 0;
            ParallelHash ph(size, t, maxIter);
            start = high_resolution_clock::now();
            ph.insertKeys(keys.data(), n, rehashesPar);
            end = high_resolution_clock::now();
            auto durationPar = duration_cast<microseconds>(end - start).count();

            printf("E1-%d t=%u exp=%u [Sequential] %8ld us, rehashes: %4u | [CUDA] %8ld us, rehashes: %4u\n", iter, t, exp, durationSeq, rehashesSeq, durationPar, rehashesPar);
        }
    }
}

void experiment2(uint32_t t)
{
    uint32_t size = pow(2, 25);
    uint32_t n = pow(2, 24);
    uint32_t maxIter = 4 * log2(n);
    vector<uint32_t> keys;
    generateRandomKeys(keys, n);
    vector<vector<uint32_t>> S(11);

    SequentialHash sh(size, t, maxIter);
    uint32_t rehashesSeq = 0;
    sh.insertKeys(keys.data(), n, rehashesSeq);

    ParallelHash ph(size, t, maxIter);
    uint32_t rehashesPar = 0;
    ph.insertKeys(keys.data(), n, rehashesPar);

    for (uint32_t i = 0; i <= 10; ++i)
    {
        for (int iter = 0; iter < 5; iter++)
        {
            generateRandomKeys(S[i], n * i / 10);
            // Fill remaining S[i] with keys randomly chosen in keys[]
            for (uint32_t j = S[i].size(); j < n; ++j)
            {
                // Pick a random index from [0, n-1]
                uint32_t idx = rand() % n;
                S[i].push_back(keys[idx]);
            }

            auto start = high_resolution_clock::now();
            for (uint32_t k = 0; k < n; ++k)
                sh.lookupKey(S[i][k]);
            auto end = high_resolution_clock::now();
            auto durationSeq = duration_cast<microseconds>(end - start).count();

            bool* results = new bool[n];
            start = high_resolution_clock::now();
            ph.lookupKeys(S[i].data(), results, n);
            end = high_resolution_clock::now();
            auto durationPar = duration_cast<microseconds>(end - start).count();
            delete [] results;

            printf("E2-%d t=%u i=%-2u [Sequential] %8ld us | [CUDA] %8ld us\n", iter, t, i, durationSeq, durationPar);
        }
    }
}

void experiment3(uint32_t t)
{
    uint32_t n = pow(2, 24);
    uint32_t maxIter = 4 * log2(n);
    vector<uint32_t> keys;
    generateRandomKeys(keys, n);

    vector<double> alpha = {1.1, 1.2, 1.3, 1.4, 1.5, 1.6, 1.7, 1.8, 1.9, 2.0, 1.01, 1.02, 1.05};

    for (double i : alpha)
    {
        for (int iter = 0; iter < 5; iter++)
        {
            uint32_t size = (uint32_t)(i * n);

            SequentialHash sh(size, t, maxIter);
            uint32_t rehashesSeq = 0;
            auto start = high_resolution_clock::now();
            sh.insertKeys(keys.data(), n, rehashesSeq);
            auto end = high_resolution_clock::now();
            auto durationSeq = duration_cast<microseconds>(end - start).count();

            ParallelHash ph(size, t, maxIter);
            uint32_t rehashesPar = 0;
            start = high_resolution_clock::now();
            ph.insertKeys(keys.data(), n, rehashesPar);
            end = high_resolution_clock::now();
            auto durationPar = duration_cast<microseconds>(end - start).count();

            printf("E3-%d t=%u size=%.2lfn [Sequential] %8ld us, rehashes: %4u | [CUDA] %8ld us, rehashes: %4u\n", iter, t, i, durationSeq, rehashesSeq, durationPar, rehashesPar);
        }
    }
}

void experiment4(uint32_t t)
{
    uint32_t n = pow(2, 24);
    uint32_t size = n * 14 / 10;
    vector<uint32_t> keys;
    generateRandomKeys(keys, n);

    int bestAlphaSeq = 0;
    long bestTimeSeq = LONG_MAX;

    int bestAlphaPar = 0;
    long bestTimePar = LONG_MAX;

    for (int i = 2; i <= 10; i++)
    {
        for (int iter = 0; iter < 5; iter++)
        {
            uint32_t maxIter = i * log2(n);

            SequentialHash sh(size, t, maxIter);
            uint32_t rehashesSeq = 0;
            auto start = high_resolution_clock::now();
            sh.insertKeys(keys.data(), n, rehashesSeq);
            auto end = high_resolution_clock::now();
            auto durationSeq = duration_cast<microseconds>(end - start).count();

            if (durationSeq < bestTimeSeq)
            {
                bestTimeSeq = durationSeq;
                bestAlphaSeq = i;
            }
            
            ParallelHash ph(size, t, maxIter);
            uint32_t rehashesPar = 0;
            start = high_resolution_clock::now();
            ph.insertKeys(keys.data(), n, rehashesPar);
            end = high_resolution_clock::now();
            auto durationPar = duration_cast<microseconds>(end - start).count();

            if (durationPar < bestTimePar)
            {
                bestTimePar = durationPar;
                bestAlphaPar = i;
            }

            printf("E4-%d t=%u maxIter=%d logn [Sequential] %8ld us, rehashes: %4u | [CUDA] %8ld us, rehashes: %4u\n", iter, t, i, durationSeq, rehashesSeq, durationPar, rehashesPar);
        }
    }

    printf ("Best alpha for Sequential: %d\n", bestAlphaSeq);
    printf ("Best alpha for CUDA: %d\n", bestAlphaPar);
}

void createHashTableFromFile(const string &filename, uint32_t t, uint32_t size, uint32_t maxIter)
{
    vector<uint32_t> keys;
    readKeysFromFile(filename, keys);
    uint32_t n = keys.size();

    uint32_t rehashesSeq = 0;
    SequentialHash sh(size, t, maxIter);
    sh.insertKeys(keys.data(), n, rehashesSeq);

    std::cout << "Final table (Sequential):" << std::endl;
    sh.printTables();

    uint32_t rehashesPar = 0;
    ParallelHash ph(size, t, maxIter);
    ph.insertKeys(keys.data(), n, rehashesPar);

    std::cout << "Final table (Parallel):" << std::endl;
    ph.printTables();
}

void searchKeysFromFile(const string &hashTableFile, const string &searchKeysFile, uint32_t t, uint32_t size, uint32_t maxIter)
{
    vector<uint32_t> hashTableKeys;
    readKeysFromFile(hashTableFile, hashTableKeys);
    uint32_t n = hashTableKeys.size();

    SequentialHash sh(size, t, maxIter);
    uint32_t rehashesSeq = 0;
    sh.insertKeys(hashTableKeys.data(), n, rehashesSeq);

    ParallelHash ph(size, t, maxIter);
    uint32_t rehashesPar = 0;
    ph.insertKeys(hashTableKeys.data(), n, rehashesPar);

    vector<uint32_t> searchKeys;
    readKeysFromFile(searchKeysFile, searchKeys);

    auto start = high_resolution_clock::now();
    for (uint32_t key : searchKeys)
        sh.lookupKey(key);
    auto end = high_resolution_clock::now();
    auto durationSeq = duration_cast<microseconds>(end - start).count();

    bool* results = new bool[searchKeys.size()];
    start = high_resolution_clock::now();
    ph.lookupKeys(searchKeys.data(), results, searchKeys.size());
    end = high_resolution_clock::now();
    auto durationPar = duration_cast<microseconds>(end - start).count();
    delete [] results;

    printf("Search keys from file [Sequential] %8ld us | [CUDA] %8ld us\n", durationSeq, durationPar);
}

int main()
{
    simpleDemo();

    printf("Experiment 1:\n");
    experiment1(2);
    experiment1(3);

    printf("Experiment 2:\n");
    experiment2(2);
    experiment2(3);

    printf("Experiment3:\n");
    experiment3(2);
    experiment3(3);

    printf("Experiment4:\n");
    experiment4(2);
    experiment4(3);

    /** Below are example usages on how to create hash table from file and search keys from file.
     * 
     * To create a hash table:
     * createHashTableFromFile("keys.txt", 2, 16, 4 * log2(16));
     * 
     * To search a set of keys from a file:
     * searchKeysFromFile("keys.txt", "search_keys.txt", 2, 16, 4 * log2(16));
     * 
     * Key file format:
     * 11231 24185 43431 5421 etc.
     * 
     **/

    return 0;
}
