#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <cmath>
#include "parallel.cuh"
#include "vanilla.hpp"

using namespace std;
using namespace chrono;

void generateRandomKeys(vector<int> &keys, int n)
{
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> dis(1, INT_MAX);

    for (int i = 0; i < n; ++i)
    {
        keys.push_back(dis(gen));
    }
}

int main()
{
    int t = 2;
    int n = 10;
    int size = 8;
    int maxIter = 4 * log2(n);
    vector<int> keys;
    generateRandomKeys(keys, n);

    cout << "Sequential Hash Table:" << endl;
    SequentialHash sh(size, t, maxIter);
    sh.insertKeys(keys.data(), n);
    sh.printTables();

    cout << "Parallel Hash Table:" << endl;
    ParallelHash ph(size, t, maxIter);
    ph.insertKeys(keys.data(), n);
    ph.printTables();

    return 0;
}
